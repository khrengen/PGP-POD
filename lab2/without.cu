#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CSC(call)                                                   \
do {                                                                \
    hipError_t res = call;                                         \
    if (res != hipSuccess) {                                       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));       \
        exit(0);                                                    \
    }                                                               \
} while(0)

double intensity2(uchar4 p) {
    return 0.299 * p.x + 0.587 * p.y + 0.114 * p.z;
}
void kernel2(uchar4 *out, uchar4 *in, int w, int h) {

    for (int y = 0; y < h; y++) {
        for (int x = 0; x < w; x++) {
            double w11 = intensity2(in[max(min(y-1,h-1),0)*w + max(min(x-1,w-1),0)]);
            double w12 = intensity2(in[max(min(y,h-1),0)*w + max(min(x-1,w-1),0)]);
            double w13 = intensity2(in[max(min(y+1,h-1),0)*w + max(min(x-1,w-1),0)]);
            double w21 = intensity2(in[max(min(y-1,h-1),0)*w + max(min(x,w-1),0)]);
            double w22 = intensity2(in[max(min(y,h-1),0)*w + max(min(x,w-1),0)]);
            double w23 = intensity2(in[max(min(y+1,h-1),0)*w + max(min(x,w-1),0)]);
            double w31 = intensity2(in[max(min(y-1,h-1),0)*w + max(min(x+1,w-1),0)]);
            double w32 = intensity2(in[max(min(y,h-1),0)*w + max(min(x+1,w-1),0)]);
            double w33 = intensity2(in[max(min(y+1,h-1),0)*w + max(min(x+1,w-1),0)]);

            double gx = w13 + 2*w23 + w33 - w11 - 2*w21 - w31;
            double gy = w31 + 2*w32 + w33 - w11 - 2*w12 - w13;

            int grad = (int)sqrt(gx*gx + gy*gy);
            int result = min(255, grad);

            out[y * w + x] = make_uchar4(result, result, result, 0);
        }
    }
}

// текстурная ссылка <тип элементов, размерность, режим нормализации>
texture<uchar4, 2, hipReadModeElementType> tex;

__device__ double intensity(uchar4 p) {
  return 0.299 * p.x + 0.587 * p.y + 0.114 * p.z;
}

__global__ void kernel(uchar4 *out, int w, int h) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    for (int y = idy; y < h; y += offsety) {
        for (int x = idx; x < w; x += offsetx) {
            double w11 = intensity(tex2D(tex, x-1, y-1));
            double w12 = intensity(tex2D(tex, x-1, y));
            double w13 = intensity(tex2D(tex, x-1, y+1));
            double w21 = intensity(tex2D(tex, x, y-1));
            double w22 = intensity(tex2D(tex, x, y));
            double w23 = intensity(tex2D(tex, x, y+1));
            double w31 = intensity(tex2D(tex, x+1, y-1));
            double w32 = intensity(tex2D(tex, x+1, y));
            double w33 = intensity(tex2D(tex, x+1, y+1));

            double gx = w13 + 2*w23 + w33 - w11 - 2*w21 - w31;
            double gy = w31 + 2*w32 + w33 - w11 - 2*w12 - w13;

            int grad = (int)sqrt(gx*gx + gy*gy);
            int result = min(255, grad);

            out[y * w + x] = make_uchar4(result, result, result, 0);
        }
    }
}

int main() {
    int w, h;
    char strIn[50];
    char strOut[50];
    scanf("%s\n%s", strIn, strOut);
    FILE *fp = fopen(strIn, "rb");
    fread(&w, sizeof(int), 1, fp);
    fread(&h, sizeof(int), 1, fp);
    uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    uchar4 *out = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    fread(data, sizeof(uchar4), w * h, fp);
    fclose(fp);

    clock_t begin = clock();
    kernel2(out, data, w,h);
    clock_t end = clock();
    printf("%f",(double)(end - begin) / CLOCKS_PER_SEC*1000000);
    // Подготовка данных для текстуры
    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));

    CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));

    // Подготовка текстурной ссылки, настройка интерфейса работы с данными
    tex.addressMode[0] = hipAddressModeClamp;  // Политика обработки выхода за границы по каждому измерению
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = ch;
    tex.filterMode = hipFilterModePoint;       // Без интерполяции при обращении по дробным координатам
    tex.normalized = false;                     // Режим нормализации координат: без нормализации

    // Связываем интерфейс с данными
    CSC(hipBindTextureToArray(tex, arr, ch));

    uchar4 *dev_out;
    CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

    kernel<<<dim3(16, 16), dim3(32, 32)>>>(dev_out, w, h);
    CSC(hipGetLastError());

    CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

    // Отвязываем данные от текстурной ссылки
    CSC(hipUnbindTexture(tex));


    CSC(hipFreeArray(arr));
    CSC(hipFree(dev_out));

    fp = fopen(strOut, "wb");
    fwrite(&w, sizeof(int), 1, fp);
    fwrite(&h, sizeof(int), 1, fp);
    fwrite(out, sizeof(uchar4), w * h, fp);
    fclose(fp);

    free(data);
    free(out);

    return 0;
}